#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define __DEBUG
#define TSCALE 1.0
#define VSQR 0.1

#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

extern int tpdt(double *t, double dt, double end_time);

/**************************************
* void __cudaSafeCall(cudaError err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = cudaThreadSynchronize();
    if( cudaSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, cudaGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}

__global__ void evolve_GPU(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t, int nThreads){
  int idx = blockIdx.x*(nThreads*nThreads)+threadIdx.x;
  if(idx<n*n){
    un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n]  + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1]) - 
                    5 * uc[idx])/(h * h) + (double)(-__expf(-TSCALE * (float)t) * pebbles[idx]));
  }
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads)
{
	hipEvent_t kstart, kstop;
	float ktime;
        
	/* HW2: Define your local variables here */
  double *uc, *uo, *nd, *cd, *od;
  double t, dt;

  //un = (double*)malloc(sizeof(double) * n * n);
  uc = (double*)malloc(sizeof(double) * n * n);
  uo = (double*)malloc(sizeof(double) * n * n);

  memcpy(uo, u0, sizeof(double) * n * n);
  memcpy(uc, u1, sizeof(double) * n * n);

  t = 0.;
  dt = h / 2.;

  /* Set up device timers */  
	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipEventCreate(&kstart));
	CUDA_CALL(hipEventCreate(&kstop));

	/* HW2: Add CUDA kernel call preperation code here */

  int threadsPerBlock = nthreads * nthreads;
  int nBlocks = (n/nthreads)*(n/nthreads);
  //dim3 blockdims(nthreads,nthreads,1);
  //dim3 griddims(n/nthreads , n/nthreads , 1 );

	/* Start GPU computation timer */
	CUDA_CALL(hipEventRecord(kstart, 0));

	/* HW2: Add main lake simulation loop here */
	CUDA_CALL(hipMalloc((void **)&od, n*n*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&cd, n*n*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&nd, n*n*sizeof(double)));

  CUDA_CALL(hipMemcpy(od,uo, n*n*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(cd,uc, n*n*sizeof(double), hipMemcpyHostToDevice));
  double *temp;
  int count=0;
  while(1){
    evolve_GPU<<<nBlocks, threadsPerBlock >>>(nd, cd, od, pebbles, n, h, dt, t, nthreads);
    temp = od;
    od = cd;
    cd = nd;
    printf(" %ld\n", t);
    if(!tpdt(&t,dt,end_time)) break;
    nd = temp;
  }
  printf("%ld,%ld",sizeof(u),u[1]);
  CUDA_CALL(hipMemcpy(u,cd, n*n*sizeof(double), hipMemcpyDeviceToHost));
        /* Stop GPU computation timer */
	CUDA_CALL(hipEventRecord(kstop, 0));
	CUDA_CALL(hipEventSynchronize(kstop));
	CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
	printf("GPU computation: %f msec\n", ktime);

	/* HW2: Add post CUDA kernel call processing and cleanup here */

	/* timer cleanup */
	CUDA_CALL(hipEventDestroy(kstart));
	CUDA_CALL(hipEventDestroy(kstop));
}

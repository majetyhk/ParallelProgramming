#include "hip/hip_runtime.h"
/*
Single Author info:
hmajety  Hari Krishna Majety
Group info:
hmajety  Hari Krishna Majety
srout Sweta Rout
mreddy2 Harshavardhan Reddy Muppidi
*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "mpi.h"
#include <unistd.h>
#define _USE_MATH_DEFINES

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1

void init(double *u, double *pebbles, int n);
void evolve(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
int tpdt(double *t, double dt, double end_time);
void print_heatmap(const char *filename, double *u, int n, double h);
void init_pebbles(double *p, int pn, int n);

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time);

extern void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads,int tag,int rank,MPI_Comm world);
char *my_itoa(int num, char *str)
{
  if(str == NULL)
  {
    return NULL;
  }
  sprintf(str, "%d", num);
  return str;
}

int main(int argc, char *argv[])
{

  if(argc != 5)
  {
    printf("Usage: %s npoints npebs time_finish nthreads \n",argv[0]);
    return 0;
  }

  int     npoints   = atoi(argv[1]);
  int     npebs     = atoi(argv[2]);
  double  end_time  = (double)atof(argv[3]);
  int     nthreads  = atoi(argv[4]);
  int 	  narea	    = npoints * npoints;

  //------------------------MPI Setup-----------------------

  int   numproc, rank;
  int tag=50;
  int startValueX, endValueX, startValueY, endValueY;             // For Defining Processes End Grid Points

  /* initialize MPI */
  MPI_Init(&argc, &argv);

  /* get the number of procs in the comm */
  MPI_Comm_size(MPI_COMM_WORLD, &numproc);

  /* get my rank in the comm */
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  /*startValueX = (rank%2) * (npoints/2);
  endValueX = (npoints/2)*(1+rank%2);

  startValueY = (rank%2)*(npoints/2);
  endValueY = (npoints/2)*(1+rank%2);*/

  startValueX = 0; // assign starting and ending indices on x and y axes in the lake for different processes based on rank
  endValueX = npoints-1;

  startValueY = rank * npoints/4; 
  endValueY = startValueY + npoints/4 -1;

  printf("Rank %d : [%d, %d] - [%d, %d]\n",rank, startValueX,endValueX,startValueY, endValueY );

  //------------------------MPI Setup-----------------------

  double *u_i0, *u_i1;
  double *u_cpu, *u_gpu, *pebs;
  double h;

  double  elapsed_gpu;
  struct timeval gpu_start, gpu_end;
  
  u_i0 = (double*)malloc(sizeof(double) * narea);
  u_i1 = (double*)malloc(sizeof(double) * narea);
  pebs = (double*)malloc(sizeof(double) * narea);

  u_cpu = (double*)malloc(sizeof(double) * narea);
  u_gpu = (double*)malloc(sizeof(double) * ((npoints/4)+2)*npoints);

  printf("Running %s with (%d x %d) grid, until %f, with %d threads\n", argv[0], npoints, npoints, end_time, nthreads);

  h = (XMAX - XMIN)/npoints;
  
  init_pebbles(pebs, npebs, npoints); // Initialize pebbles
  init(u_i0, pebs, npoints); //Initialize lake arrays
  init(u_i1, pebs, npoints);
  char fname[20];
  
  


  //----------------------MPI Manipulations--------------------------------------------

  double *uc, *un;
  int fidx,cidx; // Create indices and required arrays to pass it to kernel
  if(rank==0||rank==4){ // First and last processes will have one extra row for exchange
    uc = (double*)calloc((npoints/4 + 1)*(npoints),sizeof(double));
    un = (double*)calloc((npoints/4 + 1)*(npoints),sizeof(double));
  }
  else{ // Other processes have two extra rows, one above and one below for exchange
    uc = (double*)calloc((npoints/4 + 2)*(npoints),sizeof(double));
    un = (double*)calloc((npoints/4 + 2)*(npoints),sizeof(double));
  }
  if(rank==0){
    cidx = 0;
  }
  else{
    cidx = npoints;
  }

  // Copy the required parts of the lake into the respective arrays
  for (int j = startValueY; j < endValueY; ++j)
  {
    for (int i = startValueX; i < endValueX; ++i)
    {
      fidx = j*npoints+i;
      uc[cidx] = u_i0[fidx];
      un[cidx] = u_i1[fidx];
      cidx++;
    }
  }
  if(rank == 0) {
    //my_itoa(rank,fname);
    //strcat(fname,".dat");

    //Print initial heatmap of the lake
    print_heatmap("lake_i.dat", u_i0, npoints, h);
  }
  //print_heatmap(fname, uc, npoints, h);
  /*my_itoa(rank,fname);
  strcat(fname,".dat");
  print_heatmap(fname, uc, npoints, h);*/
  


//----------------------MPI Manipulations--------------------------------------------


  //gettimeofday(&cpu_start, NULL);
  //run_cpu(u_cpu, u_i0, u_i1, pebs, npoints, h, end_time);
  //gettimeofday(&cpu_end, NULL);

  //elapsed_cpu = ((cpu_end.tv_sec + cpu_end.tv_usec * 1e-6)-(
  //                cpu_start.tv_sec + cpu_start.tv_usec * 1e-6));
  //printf("CPU took %f seconds\n", elapsed_cpu);

  //Run the GPU code for resized arrays
  gettimeofday(&gpu_start, NULL);
  run_gpu(u_gpu, uc, un, pebs, npoints, h, end_time, nthreads,tag,rank,MPI_COMM_WORLD);  
  gettimeofday(&gpu_end, NULL);
  elapsed_gpu = ((gpu_end.tv_sec + gpu_end.tv_usec * 1e-6)-(
                  gpu_start.tv_sec + gpu_start.tv_usec * 1e-6));
  printf("GPU took %f seconds\n", elapsed_gpu);
  
  double *res;
  if(rank == 0){
    res = (double*)(malloc(npoints*npoints*sizeof(double)));
  }

  //Gather GPU Output at the the root node
  if(rank == 0){
    MPI_Gather(u_gpu,(npoints/4)*npoints,MPI_DOUBLE,res,(npoints/4)*npoints,MPI_DOUBLE,0,MPI_COMM_WORLD);
    /*for(int j = 0;j<npoints;j++){
      for(int i =0;i<npoints;i++){
        printf("[%d:%d,%d]:%lf ",rank,i,j,res[j*npoints+i]);
      }
      printf("\n");
    }*/
  }
  else{
    MPI_Gather(u_gpu+npoints,(npoints/4)*npoints,MPI_DOUBLE,res,(npoints/4)*npoints,MPI_DOUBLE,0,MPI_COMM_WORLD);
  }

  
    

  //print_heatmap("lake_f.dat", u_cpu, npoints, h);
  my_itoa(rank,fname);
  strcat(fname,"_gpu.dat");
  if(rank==0){
    //print_heatmap(fname, u_gpu, npoints, h);
    print_heatmap("lake_f_gpu.dat", res, npoints, h);
  }
  /*my_itoa(rank,fname);
  strcat(fname,"_gpu.dat");
  print_heatmap(fname, u_gpu, npoints, h);*/

  free(u_i0);
  free(u_i1);
  free(pebs);
  free(u_cpu);
  free(u_gpu);

  return 0;
}

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time)
{
  double *un, *uc, *uo;
  double t, dt;

  un = (double*)malloc(sizeof(double) * n * n);
  uc = (double*)malloc(sizeof(double) * n * n);
  uo = (double*)malloc(sizeof(double) * n * n);

  memcpy(uo, u0, sizeof(double) * n * n);
  memcpy(uc, u1, sizeof(double) * n * n);

  t = 0.;
  dt = h / 2.;

  while(1)
  {
    evolve9pt(un, uc, uo, pebbles, n, h, dt, t);

    memcpy(uo, uc, sizeof(double) * n * n);
    memcpy(uc, un, sizeof(double) * n * n);

    if(!tpdt(&t,dt,end_time)) break;
  }
  
  memcpy(u, un, sizeof(double) * n * n);
}

void init_pebbles(double *p, int pn, int n)
{
  int i, j, k, idx;
  int sz;

  srand( 2347 );
  memset(p, 0, sizeof(double) * n * n);

  for( k = 0; k < pn ; k++ )
  {
    i = rand() % (n - 4) + 2;
    j = rand() % (n - 4) + 2;
    sz = rand() % MAX_PSZ;
    idx = j + i * n;
    p[idx] = (double) sz;
  }
}

double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

int tpdt(double *t, double dt, double tf)
{
  if((*t) + dt > tf) return 0;
  (*t) = (*t) + dt;
  return 1;
}

void init(double *u, double *pebbles, int n)
{
  int i, j, idx;

  for(i = 0; i < n ; i++)
  {
    for(j = 0; j < n ; j++)
    {
      idx = j + i * n;
      u[idx] = f(pebbles[idx], 0.0);
    }
  }
}

void evolve(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t)
{
  int i, j, idx;

  for( i = 0; i < n; i++)
  {
    for( j = 0; j < n; j++)
    {
      idx = j + i * n;

      if( i == 0 || i == n - 1 || j == 0 || j == n - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n] - 4 * uc[idx])/(h * h) + f(pebbles[idx],t));
      }
    }
  }
}

void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t){
  int i, j, idx;

  for( i = 0; i < n; i++)
  {
    for( j = 0; j < n; j++)
    {
      idx = j + i * n;

      if( i == 0 || i == n - 1 || j == 0 || j == n - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n]  + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1]) - 
                    5 * uc[idx])/(h * h) + f(pebbles[idx],t));
      }
    }
  }
}

void print_heatmap(const char *filename, double *u, int n, double h)
{
  int i, j, idx;

  FILE *fp = fopen(filename, "w");  

  for( i = 0; i < n; i++ )
  {
    for( j = 0; j < n; j++ )
    {
      idx = j + i * n;
      fprintf(fp, "%lf %lf %lf\n", i*h, j*h, u[idx]*10);
    }
  }
  
  fclose(fp);
} 

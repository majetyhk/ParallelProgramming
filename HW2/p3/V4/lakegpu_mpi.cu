#include "hip/hip_runtime.h"
/*
Single Author info:
hmajety  Hari Krishna Majety
Group info:
hmajety  Hari Krishna Majety
srout Sweta Rout
mreddy2 Harshavardhan Reddy Muppidi
*/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include "mpi.h"
#define __DEBUG
#define TSCALE 1.0
#define VSQR 0.1


#define disp2darr(src,row,col) for(int kai=0;kai<row;kai++) {for(int l=0;l<col;l++) { int idx = kai*row+l; printf("%lf ",src[idx]);} printf("\n");} printf("\n")

#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

extern int tpdt(double *t, double dt, double end_time);

/**************************************
* void __cudaSafeCall(hipError_t err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}

__global__ void evolve_GPU(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t, int nThreads, int rank){
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int i, j, pebbJ, pebbIdx;
  i = idx % n;
  j = idx /n;
  pebbJ = rank*(n/4)+j;
  pebbIdx = pebbJ*n+i;
  //printf("[ %d, %d ]: %lf \n",i,j, un[idx])
  if(idx < n*((n/4)+2)){
    if(rank==0){
      //Exclude last two rows in the array for computations apart from the top row of the lake
      if( i == 0 || i == n-1|| j == 0 || j >= n/4)
      {
        un[idx] = 0.;
        //printf("[ %d, %d ]: %lf \n",i,j, un[idx]);
      }
      else{
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n]  + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1]) - 
                    5 * uc[idx])/(h * h) + (double)(-__expf(-TSCALE * (float)t) * pebbles[pebbIdx]));
        //printf("[%d: %d, %d -(%d)] - %lf \n",rank,i,j, idx,un[idx]);
      }
    }
    else{
      //Exclude first and last row in the array for computations
      if( i  == 0 || i == n-1|| j == 0 || j >= n/4 + 1)
      {
        un[idx] = 0.;
      }
      else{
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n]  + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1]) - 
                    5 * uc[idx])/(h * h) + (double)(-__expf(-TSCALE * (float)t) * pebbles[pebbIdx]));
        //printf("[%d: %d, %d -(%d)] - %lf \n",rank,i,j, idx,un[idx]);
      }
    }
    /*else if(rank == 3){
      //Exclude first two rows in the array for computations apart from the last row of the lake
      if( i == 0 || i == n-1|| j == 0  || j >= n/4+1)
      {
        un[idx] = 0.;
      }
      else{
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n]  + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1]) - 
                    5 * uc[idx])/(h * h) + (double)(-__expf(-TSCALE * (float)t) * pebbles[idx]));
        printf("[%d: %d, %d -(%d)] - %lf \n",rank,i,j, idx,un[idx]);
      }
    }*/
    
    //printf("[%d: %d, %d -(%d)] - %lf \n",rank,i,j, idx,un[idx]);
    //fflush(stdin);
    
    //fflush(stdin);
  }
 
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads,int tag, int rank, MPI_Comm world)
{
  hipEvent_t kstart, kstop;
  float ktime;
  int npoints = n;
  MPI_Status status;
  /* HW2: Define your local variables here */
  double *uc, *uo, *nd, *cd, *od, *pebblesd;
  double t, dt;

  //un = (double*)malloc(sizeof(double) * n * n);
  uc = (double*)calloc( ((n/4)+2) * n,sizeof(double));
  uo = (double*)calloc( ((n/4)+2) * n,sizeof(double));


  memcpy(uo, u0, sizeof(double) * ((n/4)+2) * n);
  memcpy(uc, u1, sizeof(double) * ((n/4)+2) * n);
   

  t = 0.0;
  dt = h / 2.;
  /*for (int i = 0; i < n*n; ++i)
  {
    printf("%d - %lf; ",i, pebbles[i]);
  }*/
  /* Set up device timers */  
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipEventCreate(&kstart));
  CUDA_CALL(hipEventCreate(&kstop));

  /* HW2: Add CUDA kernel call preperation code here */

  int threadsPerBlock = nthreads * nthreads;
  int nBlocks = (n/nthreads)*(n/nthreads);
  //dim3 blockdims(nthreads,nthreads,1);
  //dim3 griddims(n/nthreads , n/nthreads , 1 );

  /* Start GPU computation timer */
  CUDA_CALL(hipEventRecord(kstart, 0));

  /* HW2: Add main lake simulation loop here */
  CUDA_CALL(hipMalloc((void **)&od, ((n/4)+2)*n*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&cd, ((n/4)+2)*n*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&nd, ((n/4)+2)*n*sizeof(double)));
  CUDA_CALL(hipMemset(od,0,((n/4)+2)*n*sizeof(double)));
  CUDA_CALL(hipMemset(cd,0,((n/4)+2)*n*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&pebblesd, n*n*sizeof(double)));

  //CUDA_CALL(hipMemcpy(od,uo, ((n/4)+2)*((n/4)+2)*sizeof(double), hipMemcpyHostToDevice));
  //CUDA_CALL(hipMemcpy(cd,uc, ((n/4)+2)*((n/4)+2)*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(pebblesd,pebbles, n*n*sizeof(double), hipMemcpyHostToDevice));

  printf("%d rank\n",rank);
  //int count=0;
  while(1){
    //Transfer the boundary slices to other processes for computation
    if(rank==0){
      tag = rank;
      //Stage 1
      MPI_Send(uo+(npoints/4 - 1)*npoints,npoints,MPI_DOUBLE,rank+1,tag,world);
      MPI_Send(uc+(npoints/4 - 1)*npoints,npoints,MPI_DOUBLE,rank+1,tag,world);
      //Stage 2
      //-----No actions for rank 0 in this stage------
      //Stage 3
      MPI_Recv(uo+(npoints/4)*npoints,npoints, MPI_DOUBLE, rank+1, tag, world, &status);
      MPI_Recv(uc+(npoints/4)*npoints,npoints, MPI_DOUBLE, rank+1, tag, world, &status);

    }
    else if(rank == 1){
      tag = rank-1;
      //Stage 1
      MPI_Recv(uo,npoints,MPI_DOUBLE,rank-1,tag,world, &status);
      MPI_Recv(uc,npoints,MPI_DOUBLE,rank-1,tag,world, &status);
      //Stage 2
      MPI_Send(uo+(npoints/4)*npoints,npoints,MPI_DOUBLE,rank+1,rank+1,world);
      MPI_Send(uc+(npoints/4)*npoints,npoints,MPI_DOUBLE,rank+1,rank+1,world);
      //Stage 3
      MPI_Send(uo+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);
      MPI_Send(uc+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);
      //Stage 4
      MPI_Recv(uo+(npoints/4 + 1)*npoints,npoints, MPI_DOUBLE, rank+1, rank+1, world, &status);
      MPI_Recv(uc+(npoints/4 + 1)*npoints,npoints, MPI_DOUBLE, rank+1, rank+1, world, &status);

    }
    else if(rank==2){
      tag = rank;
      //Stage 1
      MPI_Send(uo+(npoints/4)*npoints,npoints,MPI_DOUBLE,rank+1,tag,world);
      MPI_Send(uc+(npoints/4)*npoints,npoints,MPI_DOUBLE,rank+1,tag,world);
      //Stage 2
      MPI_Recv(uo,npoints,MPI_DOUBLE,rank-1,tag,world, &status);
      MPI_Recv(uc,npoints,MPI_DOUBLE,rank-1,tag,world, &status);
      //Stage 3
      MPI_Recv(uo+(npoints/4 + 1)*npoints,npoints, MPI_DOUBLE, rank+1, tag,world, &status);
      MPI_Recv(uc+(npoints/4 + 1)*npoints,npoints, MPI_DOUBLE, rank+1, tag, world, &status);
      //Stage 4
      MPI_Send(uo+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);
      MPI_Send(uc+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);
    }
    else{
      tag = rank-1;
      //Stage 1
      MPI_Recv(uo, npoints, MPI_DOUBLE, rank-1,tag, world, &status);
      MPI_Recv(uc, npoints, MPI_DOUBLE, rank-1, tag, world, &status);
      //Stage 2
      //------ No actions for rank 3 in this stage
      //Stage 3
      MPI_Send(uo+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);
      MPI_Send(uc+npoints,npoints,MPI_DOUBLE,rank-1,tag,world);

    }
    //disp2darr(uc,npoints/4+1,npoints);
    /*if(rank==0||rank==3){
      for(int j = 0;j<npoints/4+1;j++){
        for(int i =0;i<npoints;i++){
          printf("[%d:%d,%d]:%lf ",rank,i,j,uc[j*npoints+i]);
        }
        printf("\n");
      }
    }
    else{
      for(int j = 0;j<npoints/4+2;j++){
        for(int i =0;i<npoints;i++){
          printf("[%d:%d,%d]:%lf ",rank,i,j,uc[j*npoints+i]);
        }
        printf("\n");
      }
    }*/
    
    CUDA_CALL(hipMemcpy(od,uo, ((n/4)+2)*n*sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(cd,uc, ((n/4)+2)*n*sizeof(double), hipMemcpyHostToDevice));
    //sleep(rank*0.6);
    //Invoke Cuda Kernel with same number of threads in all nodes and handle the additional threads within the Kernel
    evolve_GPU<<<nBlocks,threadsPerBlock>>>(nd,cd,od,pebblesd,n,h,dt,t,nthreads, rank);

    hipMemcpy(u,nd, ((n/4)+2)*n*sizeof(double), hipMemcpyDeviceToHost);
    memcpy(uo, uc, sizeof(double) * ((n/4)+2) * n);
    memcpy(uc, u, sizeof(double) * ((n/4)+2) * n);
  	if(!tpdt(&t,dt,end_time)) break;
    //count++;
    //if(count==1) break;

  }
  
  /*if(rank==0||rank==3){
      for(int j = 0;j<npoints/4+1;j++){
        for(int i =0;i<npoints;i++){
          printf("[%d:%d,%d]:%lf ",rank,i,j,u[j*npoints+i]);
        }
        printf("\n");
      }
    }
    else{
      for(int j = 0;j<npoints/4+2;j++){
        for(int i =0;i<npoints;i++){
          printf("[%d:%d,%d]:%lf ",rank,i,j,u[j*npoints+i]);
        }
        printf("\n");
      }
    }*/
  /* Stop GPU computation timer */
  /*for (int i = 0; i < n;  ++i)
  {
    fflush(stdin);
    for (int j = 0; j < n/4; ++j)
    {
      printf("%d:%lf ", u[i*n+j]);
    }
    printf("\n");
  }*/
  CUDA_CALL(hipEventRecord(kstop, 0));
  CUDA_CALL(hipEventSynchronize(kstop));
  CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
  printf("GPU computation: %f msec\n", ktime);

  /* HW2: Add post CUDA kernel call processing and cleanup here */
  CUDA_CALL(hipFree(od));
  CUDA_CALL(hipFree(cd));
  CUDA_CALL(hipFree(nd));
  CUDA_CALL(hipFree(pebblesd));
  free(uc);
  free(uo);
  /* timer cleanup */
  CUDA_CALL(hipEventDestroy(kstart));
  CUDA_CALL(hipEventDestroy(kstop));
  CUDA_CALL(hipDeviceReset());
}

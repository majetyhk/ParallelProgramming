#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<hiprand/hiprand_kernel.h>

#define SEED 35791246

__global__ void setup_kernel(hiprandState *state, int numElements)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    if(id<numElements){
    	hiprand_init(SEED, id, 0, &state[id]);
    }
    
}


__global__ void getMonteCarloVal(float *dX,float *dY, float *dZ, hiprandState *state, int numElements){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx<numElements){
		//dX[idx] = rand()/(float)RAND_MAX;
		//dY[idx] = rand()/(float)RAND_MAX;
	    //curandGenerator_t prng;
	    //curandCreateGenerator(&prng, CURAND_RNG_PSEUDO_DEFAULT);
	    //curandSetPseudoRandomGeneratorSeed(prng, SEED);
		//curandGenerateUniform(prng, dX+idx, 1);
		//curandGenerateUniform(prng, dY+idx, 1);
		hiprandState localState = state[idx];
		dX[idx] = hiprand_uniform(&localState);
		dY[idx] = hiprand_uniform(&localState);
		dZ[idx] = ((dX[idx]*dX[idx] + dY[idx]*dY[idx])<=1?1:0);
		state[idx] = localState;
		//curandDestroyGenerator(prng);
	}
}


int main(int argc, char** argv){
	//cudaError_t err = cudaSuccess;

	int numElements = atoi(argv[1]);
	size_t size = numElements * sizeof(float);
	float *dX, *dY, *dZ, *hZ;
	hipMalloc((void **)&dX, size);
	hipMalloc((void **)&dY, size);
	hipMalloc((void **)&dZ, size);
	hZ = (float *)malloc(size);
	hiprandState *devStates;
	hipMalloc((void **)&devStates, numElements * sizeof(hiprandState));
	//Cuda Configuration
	int threadsPerBlock = 256;
	int nBlocks = (numElements+threadsPerBlock-1)/threadsPerBlock;
	setup_kernel<<<nBlocks,threadsPerBlock>>>(devStates,numElements);
	getMonteCarloVal<<<nBlocks,threadsPerBlock>>>(dX, dY, dZ, devStates, numElements);
	hipMemcpy(hZ, dZ, numElements, hipMemcpyDeviceToHost);
	int count = 0;
	for (int i = 0; i < numElements; ++i)
	{
		count=count+hZ[i];
		printf("%f\n", hZ[i]);
	}
	printf("%d\n",count);
	printf("The approximate value of Pi is %f\n", ((float)count/numElements)*4 );
	hipFree(dX);
	hipFree(dY);
	hipFree(dZ);
	free(hZ);
	hipDeviceReset();
	return 0;
}
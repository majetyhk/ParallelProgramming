/*
Single Author info:
hmajety  Hari Krishna Majety
Group info:
hmajety  Hari Krishna Majety
srout Sweta Rout
mreddy2 Harshavardhan Reddy Muppidi
*/
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<hiprand/hiprand_kernel.h>

#define SEED 35791246

__global__ void setup_kernel(hiprandState *state, int numElements)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    if(id<numElements){
    	hiprand_init(SEED, id, 0, &state[id]);
    }
    
}


__global__ void getMonteCarloVal(float *dX,float *dY, float *dZ, hiprandState *state, int numElements){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	//Ignore additional threads spawned
	if(idx<numElements){
		//dX[idx] = rand()/(float)RAND_MAX;
		//dY[idx] = rand()/(float)RAND_MAX;
	    //curandGenerator_t prng;
	    //curandCreateGenerator(&prng, CURAND_RNG_PSEUDO_DEFAULT);
	    //curandSetPseudoRandomGeneratorSeed(prng, SEED);
		//curandGenerateUniform(prng, dX+idx, 1);
		//curandGenerateUniform(prng, dY+idx, 1);
		hiprandState localState = state[idx];
		dX[idx] = hiprand_uniform(&localState);
		dY[idx] = hiprand_uniform(&localState);
		dZ[idx] = ((dX[idx]*dX[idx] + dY[idx]*dY[idx])<=1?1:0);
		state[idx] = localState;
		//curandDestroyGenerator(prng);
	}
}


int main(int argc, char** argv){
	//cudaError_t err = cudaSuccess;

	int numElements = atoi(argv[1]);
	size_t size = numElements * sizeof(float);
	float *dX, *dY, *dZ, *hZ;
	//float *hX. *hY;
	hipMalloc((void **)&dX, size);
	hipMalloc((void **)&dY, size);
	hipMalloc((void **)&dZ, size);
	hZ = (float *)malloc(size);
	//hX = (float *)malloc(size);
	//hY = (float *)malloc(size);
	hiprandState *devStates;
	hipMalloc((void **)&devStates, numElements * sizeof(hiprandState));
	//Cuda Configuration
	int threadsPerBlock = 256;
	int nBlocks = (numElements+threadsPerBlock-1)/threadsPerBlock;
	printf("%d\n", nBlocks);
	//Setup Curand generator states
	setup_kernel<<<nBlocks,threadsPerBlock>>>(devStates,numElements);
	//Call the Cuda kernel to perform Monte Carlo simulation
	getMonteCarloVal<<<nBlocks,threadsPerBlock>>>(dX, dY, dZ, devStates, numElements);
	hipMemcpy(hZ, dZ, numElements*sizeof(float), hipMemcpyDeviceToHost);
	//cudaMemcpy(hX, dX, numElements*sizeof(float), cudaMemcpyDeviceToHost);
	//cudaMemcpy(hY, dY, numElements*sizeof(float), cudaMemcpyDeviceToHost);
	int count = 0;
	//Aggregate the values received from the GPU
	for (int i = 0; i < numElements; ++i)
	{
		count=count+hZ[i];
		//printf("%f,%f - %f\n",hX[i],hY[i],hZ[i]);
	}
	//printf("%d\n",count);
	printf("The approximate value of Pi is %f\n", ((float)count/numElements)*4 );
	hipFree(dX);
	//cudaFree(dY);
	//cudaFree(dZ);
	free(hZ);
	hipDeviceReset();
	return 0;
}